#include "hip/hip_runtime.h"
#include "SandCudaUtils.h"
namespace VPE {
__global__ void Kernel_UpdateParticleBuffer(size_t particle_count,
                                            int split_num,
                                            float pos_offset_x,
                                            float pos_offset_y,
                                            float pos_offset_z,
                                            float *dst_buffer_d,
                                            float *position_d,
                                            float *radius_d) {
    size_t index = threadIdx.x + (blockIdx.x * blockDim.x);

    if (index >= particle_count) {
        return;
    }

    float *pos = position_d + index * 3;
    for (int i = 0; i < split_num; i++) {
        int target_index = index * split_num + i;
        float r = radius_d[target_index];
        float *dst = dst_buffer_d + 4 * target_index;
        dst[0] = static_cast<float>(pos[0] + pos_offset_x);
        // offset by radius to make rendering result less flat
        dst[1] = static_cast<float>(pos[1] * 0.5 + pos_offset_y + r);
        dst[2] = static_cast<float>(pos[2] + pos_offset_z);
        dst[3] = r;
    }
}

void UpdateParticleBufferCuda(size_t particle_count,
                              int split_num,
                              float pos_offset_x,
                              float pos_offset_y,
                              float pos_offset_z,
                              float *dst_buffer_d,
                              float *position_d,
                              float *radius_d) {
    Kernel_UpdateParticleBuffer <<< (particle_count + 511) / 512, 512>>>(
        particle_count,
        split_num,
        pos_offset_x, pos_offset_y, pos_offset_z,
        dst_buffer_d,
        position_d,
        radius_d);
}

__global__ void Kernal_ConvertDoubleToFloat(double *src,
                                            int src_pitch,
                                            float *dst,
                                            int count_x, int count_y) {
    size_t x = threadIdx.x + (blockIdx.x * blockDim.x);
    size_t y = threadIdx.y + (blockIdx.y * blockDim.y);

    if (x >= count_x || y >= count_y) {
        return;
    }
    dst[x + y * count_x] = static_cast<float>(src[x + y * src_pitch]);
}

void CopyDoubleToFloatCuda(double *src, int src_pitch, float *dst, int count_x, int count_y) {
    dim3 dimBlock(16, 16);
    dim3 dimGrid;
    dimGrid.x = (count_x + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (count_y + dimBlock.y - 1) / dimBlock.y;
    Kernal_ConvertDoubleToFloat<<<dimGrid, dimBlock>>>(src, src_pitch, dst, count_x, count_y);
}
}  // namespace VPE